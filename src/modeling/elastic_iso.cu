#include "hip/hip_runtime.h"
# include "elastic_iso.cuh"

void Elastic_ISO::set_specifications()
{
    modeling_type = "elastic_iso";
    modeling_name = "Modeling type: Elastic isotropic solver";

    hipMalloc((void**)&(d_skw), DGS*DGS*sizeof(float));
    
    hipMalloc((void**)&(d_rkwPs), DGS*DGS*max_spread*sizeof(float));
    hipMalloc((void**)&(d_rkwVx), DGS*DGS*max_spread*sizeof(float));
    hipMalloc((void**)&(d_rkwVz), DGS*DGS*max_spread*sizeof(float));

    auto * Cij = new float[nPoints]();

    std::string vp_file = catch_parameter("vp_model_file", parameters);
    std::string ro_file = catch_parameter("ro_model_file", parameters);
    std::string Cijkl_folder = catch_parameter("Cijkl_folder", parameters);

    float * S = new float[matsize]();
    import_binary_float(vp_file, Cij, nPoints);
    expand_boundary(Cij, S);

    # pragma omp parallel for
    for (int index = 0; index < matsize; index++)
        S[index] = 1.0f / S[index];

    hipMalloc((void**)&(d_S), matsize*sizeof(float));
    hipMemcpy(d_S, S, matsize*sizeof(float), hipMemcpyHostToDevice);
    delete[] S;

    auto * B = new float[matsize]();
    auto * uB = new uintc[matsize]();
    import_binary_float(ro_file, Cij, nPoints);
    expand_boundary(Cij, B);

    # pragma omp parallel for
    for (int index = 0; index < matsize; index++)
        B[index] = 1.0f / B[index];

    compression(B, uB, matsize, maxB, minB);    
    hipMalloc((void**)&(d_B), matsize*sizeof(uintc));
    hipMemcpy(d_B, uB, matsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] B;
    delete[] uB;

    auto * C13 = new float[matsize]();
    auto * uC13 = new uintc[matsize]();
    import_binary_float(Cijkl_folder + "C13.bin", Cij, nPoints);
    expand_boundary(Cij, C13);
    compression(C13, uC13, matsize, maxC13, minC13);    
    hipMalloc((void**)&(d_C13), matsize*sizeof(uintc));
    hipMemcpy(d_C13, uC13, matsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] C13;
    delete[] uC13;

    auto * C55 = new float[matsize]();
    auto * uC55 = new uintc[matsize]();
    import_binary_float(Cijkl_folder + "C55.bin", Cij, nPoints);
    expand_boundary(Cij, C55);
    compression(C55, uC55, matsize, maxC55, minC55);    
    hipMalloc((void**)&(d_C55), matsize*sizeof(uintc));
    hipMemcpy(d_C55, uC55, matsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] C55;
    delete[] uC55;
}

void Elastic_ISO::initialization()
{
    float beta = 5.0f;

    sx = geometry->xsrc[geometry->sInd[srcId]];
    sz = geometry->zsrc[geometry->sInd[srcId]];

    sIdx = (int)((sx + 0.5f*dx) / dx);
    sIdz = (int)((sz + 0.5f*dz) / dz);

    float * h_skw = new float[DGS*DGS]();

    auto skw = kaiser_weights(sx, sz, sIdx, sIdz, dx, dz, beta);

    for (int zId = 0; zId < DGS; zId++)
        for (int xId = 0; xId < DGS; xId++)
            h_skw[zId + xId*DGS] = skw[zId][xId];

    sIdx += nb; 
    sIdz += nb;

    int * h_rIdx = new int[max_spread]();
    int * h_rIdz = new int[max_spread]();

    float * h_rkwPs = new float[DGS*DGS*max_spread]();
    float * h_rkwVx = new float[DGS*DGS*max_spread]();
    float * h_rkwVz = new float[DGS*DGS*max_spread]();

    int spreadId = 0;

    for (recId = geometry->iRec[srcId]; recId < geometry->fRec[srcId]; recId++)
    {
        float rx = geometry->xrec[recId];
        float rz = geometry->zrec[recId];
        
        int rIdx = (int)((rx + 0.5f*dz) / dx);
        int rIdz = (int)((rz + 0.5f*dz) / dz);
    
        auto rkwPs = kaiser_weights(rx, rz, rIdx, rIdz, dx, dz, beta);
        auto rkwVx = kaiser_weights(rx + 0.5f*dx, rz, rIdx, rIdz, dx, dz, beta);
        auto rkwVz = kaiser_weights(rx, rz + 0.5f*dz, rIdx, rIdz, dx, dz, beta);
        
        for (int zId = 0; zId < DGS; zId++)
        {
            for (int xId = 0; xId < DGS; xId++)
            {
                h_rkwPs[zId + xId*DGS + spreadId*DGS*DGS] = rkwPs[zId][xId];
                h_rkwVx[zId + xId*DGS + spreadId*DGS*DGS] = rkwVx[zId][xId];
                h_rkwVz[zId + xId*DGS + spreadId*DGS*DGS] = rkwVz[zId][xId];
            }
        }

        h_rIdx[spreadId] = rIdx + nb;
        h_rIdz[spreadId] = rIdz + nb;

        ++spreadId;
    }

    hipMemcpy(d_skw, h_skw, DGS*DGS*sizeof(float), hipMemcpyHostToDevice);
    
    hipMemcpy(d_rkwPs, h_rkwPs, DGS*DGS*max_spread*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rkwVx, h_rkwVx, DGS*DGS*max_spread*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rkwVz, h_rkwVz, DGS*DGS*max_spread*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_rIdx, h_rIdx, max_spread*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rIdz, h_rIdz, max_spread*sizeof(int), hipMemcpyHostToDevice);

    delete[] h_skw;
    delete[] h_rkwPs;
    delete[] h_rkwVx;
    delete[] h_rkwVz;
    delete[] h_rIdx;
    delete[] h_rIdz;
}

void Elastic_ISO::compute_eikonal()
{
    dim3 grid(1,1,1);
    dim3 block(MESHDIM+1,MESHDIM+1,1);

    time_set<<<nBlocks,NTHREADS>>>(d_T, matsize);
    time_init<<<grid,block>>>(d_T,d_S,sx,sz,dx,dz,sIdx,sIdz,nzz,nb);
    eikonal_solver();
}

void Elastic_ISO::compute_velocity()
{
    compute_velocity_ssg<<<nBlocks,NTHREADS>>>(d_Vx, d_Vz, d_Txx, d_Tzz, d_Txz, d_T, d_B, maxB, minB, d1D, d2D, 
                                               d_wavelet, dx, dz, dt, timeId, tlag, sIdx, sIdz, d_skw, nxx, nzz, nb, nt);
}

void Elastic_ISO::compute_pressure()
{
    compute_pressure_ssg<<<nBlocks,NTHREADS>>>(d_Vx, d_Vz, d_Txx, d_Tzz, d_Txz, d_P, d_T, d_C55, d_C13, maxC55, 
                                               minC55, maxC13, minC13, timeId, tlag, dx, dz, dt, nxx, nzz);    
}

__global__ void compute_velocity_ssg(float * Vx, float * Vz, float * Txx, float * Tzz, float * Txz, float * T, uintc * B, float maxB, float minB, 
                                     float * damp1D, float * damp2D, float * wavelet, float dx, float dz, float dt, int tId, int tlag, int sIdx, 
                                     int sIdz, float * skw, int nxx, int nzz, int nb, int nt)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i = (int)(index % nzz);
    int j = (int)(index / nzz);

    float Bn, Bm;

    if ((index == 0) && (tId < nt))
    {
        for (int i = 0; i < DGS; i++)
        {
            int zi = sIdz + i - 2;
            for (int j = 0; j < DGS; j++)
            {
                int xi = sIdx + j - 2;

                Txx[zi + xi*nzz] += skw[i + j*DGS]*wavelet[tId] / (dx*dz);
                Tzz[zi + xi*nzz] += skw[i + j*DGS]*wavelet[tId] / (dx*dz);
            }
        }
    }

    if ((T[index] < (float)(tId + tlag)*dt) && (index < nxx*nzz))
    {
        Bn = (minB + (static_cast<float>(B[index]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));

        if((i >= 3) && (i < nzz-4) && (j > 3) && (j < nxx-3)) 
        {
            float dTxx_dx = (FDM1*(Txx[i + (j-4)*nzz] - Txx[i + (j+3)*nzz]) +
                             FDM2*(Txx[i + (j+2)*nzz] - Txx[i + (j-3)*nzz]) +
                             FDM3*(Txx[i + (j-2)*nzz] - Txx[i + (j+1)*nzz]) +
                             FDM4*(Txx[i + j*nzz]     - Txx[i + (j-1)*nzz])) / dx;

            float dTxz_dz = (FDM1*(Txz[(i-3) + j*nzz] - Txz[(i+4) + j*nzz]) +
                             FDM2*(Txz[(i+3) + j*nzz] - Txz[(i-2) + j*nzz]) +
                             FDM3*(Txz[(i-1) + j*nzz] - Txz[(i+2) + j*nzz]) +
                             FDM4*(Txz[(i+1) + j*nzz] - Txz[i + j*nzz])) / dz;

            Bm = (minB + (static_cast<float>(B[i + (j+1)*nzz]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));

            float Bx = 0.5f*(Bn + Bm);

            Vx[index] += dt*Bx*(dTxx_dx + dTxz_dz); 
        }

        if((i > 3) && (i < nzz-3) && (j >= 3) && (j < nxx-4)) 
        {
            float dTxz_dx = (FDM1*(Txz[i + (j-3)*nzz] - Txz[i + (j+4)*nzz]) +
                             FDM2*(Txz[i + (j+3)*nzz] - Txz[i + (j-2)*nzz]) +
                             FDM3*(Txz[i + (j-1)*nzz] - Txz[i + (j+2)*nzz]) +
                             FDM4*(Txz[i + (j+1)*nzz] - Txz[i + j*nzz])) / dx;

            float dTzz_dz = (FDM1*(Tzz[(i-4) + j*nzz] - Tzz[(i+3) + j*nzz]) +
                             FDM2*(Tzz[(i+2) + j*nzz] - Tzz[(i-3) + j*nzz]) +
                             FDM3*(Tzz[(i-2) + j*nzz] - Tzz[(i+1) + j*nzz]) +
                             FDM4*(Tzz[i + j*nzz]     - Tzz[(i-1) + j*nzz])) / dz;

            Bm = (minB + (static_cast<float>(B[(i+1) + j*nzz]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));

            float Bz = 0.5f*(Bn + Bm);

            Vz[index] += dt*Bz*(dTxz_dx + dTzz_dz); 
        }

    	float damper = get_boundary_damper(damp1D, damp2D, i, j, nxx, nzz, nb);

        Vx[index] *= damper;
        Vz[index] *= damper;

        Txx[index] *= damper;
        Tzz[index] *= damper;
        Txz[index] *= damper;
    }
}

__global__ void compute_pressure_ssg(float * Vx, float * Vz, float * Txx, float * Tzz, float * Txz, float * P, float * T, uintc * C44, uintc * C13, float maxC55, 
                                     float minC55, float maxC13, float minC13, int tId, int tlag, float dx, float dz, float dt, int nxx, int nzz)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i = (int)(index % nzz);
    int j = (int)(index / nzz);

    float c44_1, c44_2, c44_3, c44_4;

    if ((T[index] < (float)(tId + tlag)*dt) && (index < nxx*nzz))
    {
        if((i >= 3) && (i < nzz-4) && (j >= 3) && (j < nxx-4)) 
        {    
            float dVx_dx = (FDM1*(Vx[i + (j-3)*nzz] - Vx[i + (j+4)*nzz]) +
                            FDM2*(Vx[i + (j+3)*nzz] - Vx[i + (j-2)*nzz]) +
                            FDM3*(Vx[i + (j-1)*nzz] - Vx[i + (j+2)*nzz]) +
                            FDM4*(Vx[i + (j+1)*nzz] - Vx[i + j*nzz])) / dx;

            float dVz_dz = (FDM1*(Vz[(i-3) + j*nzz] - Vz[(i+4) + j*nzz]) +
                            FDM2*(Vz[(i+3) + j*nzz] - Vz[(i-2) + j*nzz]) +
                            FDM3*(Vz[(i-1) + j*nzz] - Vz[(i+2) + j*nzz]) +
                            FDM4*(Vz[(i+1) + j*nzz] - Vz[i + j*nzz])) / dz;
            
            float c13 = (minC13 + (static_cast<float>(C13[index]) - 1.0f) * (maxC13 - minC13) / (COMPRESS - 1));
            float c44 = (minC55 + (static_cast<float>(C44[index]) - 1.0f) * (maxC55 - minC55) / (COMPRESS - 1));

            Txx[index] += dt*((c13 + 2*c44)*dVx_dx + c13*dVz_dz);
            Tzz[index] += dt*((c13 + 2*c44)*dVz_dz + c13*dVx_dx);                    
        }

        if((i > 3) && (i < nzz-3) && (j > 3) && (j < nxx-3)) 
        {
            float dVx_dz = (FDM1*(Vx[(i-4) + j*nzz] - Vx[(i+3) + j*nzz]) +
                            FDM2*(Vx[(i+2) + j*nzz] - Vx[(i-3) + j*nzz]) +
                            FDM3*(Vx[(i-2) + j*nzz] - Vx[(i+1) + j*nzz]) +
                            FDM4*(Vx[i + j*nzz]     - Vx[(i-1) + j*nzz])) / dz;

            float dVz_dx = (FDM1*(Vz[i + (j-4)*nzz] - Vz[i + (j+3)*nzz]) +
                            FDM2*(Vz[i + (j+2)*nzz] - Vz[i + (j-3)*nzz]) +
                            FDM3*(Vz[i + (j-2)*nzz] - Vz[i + (j+1)*nzz]) +
                            FDM4*(Vz[i + j*nzz]     - Vz[i + (j-1)*nzz])) / dx;

            c44_1 = (minC55 + (static_cast<float>(C44[(i+1) + (j+1)*nzz]) - 1.0f) * (maxC55 - minC55) / (COMPRESS - 1));
            c44_2 = (minC55 + (static_cast<float>(C44[i + (j+1)*nzz]) - 1.0f) * (maxC55 - minC55) / (COMPRESS - 1));
            c44_3 = (minC55 + (static_cast<float>(C44[(i+1) + j*nzz]) - 1.0f) * (maxC55 - minC55) / (COMPRESS - 1));
            c44_4 = (minC55 + (static_cast<float>(C44[i + j*nzz]) - 1.0f) * (maxC55 - minC55) / (COMPRESS - 1));

            float Mxz = powf(0.25f*(1.0f/c44_1 + 1.0f/c44_2 + 1.0f/c44_3 + 1.0f/c44_4),-1.0f);

            Txz[index] += dt*Mxz*(dVx_dz + dVz_dx);
        }

        if ((i > 3) && (i < nzz-4) && (j > 3) && (j < nxx-4))
            P[index] = 0.5f*(Txx[index] + Tzz[index]);
    }
}