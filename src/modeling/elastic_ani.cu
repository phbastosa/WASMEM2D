#include "hip/hip_runtime.h"
# include "elastic_ani.cuh"

void Elastic_ANI::set_specifications()
{
    modeling_type = "elastic_ani";
    modeling_name = "Modeling type: Elastic anisotropic solver";

    hipMalloc((void**)&(d_skw), DGS*DGS*sizeof(float));
    
    hipMalloc((void**)&(d_rkwPs), DGS*DGS*max_spread*sizeof(float));
    hipMalloc((void**)&(d_rkwVx), DGS*DGS*max_spread*sizeof(float));
    hipMalloc((void**)&(d_rkwVz), DGS*DGS*max_spread*sizeof(float));

    auto * Cij = new float[nPoints]();

    std::string vp_file = catch_parameter("vp_model_file", parameters);
    std::string ro_file = catch_parameter("ro_model_file", parameters);
    std::string Cijkl_folder = catch_parameter("Cijkl_folder", parameters);

    float * S = new float[matsize]();
    import_binary_float(vp_file, Cij, nPoints);
    expand_boundary(Cij, S);

    # pragma omp parallel for
    for (int index = 0; index < matsize; index++)
        S[index] = 1.0f / S[index];

    hipMalloc((void**)&(d_S), matsize*sizeof(float));
    hipMemcpy(d_S, S, matsize*sizeof(float), hipMemcpyHostToDevice);

    auto * B = new float[matsize]();
    auto * uB = new uintc[matsize]();
    import_binary_float(ro_file, Cij, nPoints);
    expand_boundary(Cij, B);

    # pragma omp parallel for
    for (int index = 0; index < matsize; index++)
        B[index] = 1.0f / B[index];

    compression(B, uB, matsize, maxB, minB);    
    hipMalloc((void**)&(d_B), matsize*sizeof(uintc));
    hipMemcpy(d_B, uB, matsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] uB;

    auto * C11 = new float[matsize]();
    auto * uC11 = new uintc[matsize]();
    import_binary_float(Cijkl_folder + "C11.bin", Cij, nPoints);
    expand_boundary(Cij, C11);
    compression(C11, uC11, matsize, maxC11, minC11);        
    hipMalloc((void**)&(d_C11), matsize*sizeof(uintc));
    hipMemcpy(d_C11, uC11, matsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] C11;
    delete[] uC11;

    auto * C13 = new float[matsize]();
    auto * uC13 = new uintc[matsize]();
    import_binary_float(Cijkl_folder + "C13.bin", Cij, nPoints);
    expand_boundary(Cij, C13);
    compression(C13, uC13, matsize, maxC13, minC13);    
    hipMalloc((void**)&(d_C13), matsize*sizeof(uintc));
    hipMemcpy(d_C13, uC13, matsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] C13;
    delete[] uC13;

    auto * C15 = new float[matsize]();
    auto * uC15 = new uintc[matsize]();
    import_binary_float(Cijkl_folder + "C15.bin", Cij, nPoints);
    expand_boundary(Cij, C15);
    compression(C15, uC15, matsize, maxC15, minC15);    
    hipMalloc((void**)&(d_C15), matsize*sizeof(uintc));
    hipMemcpy(d_C15, uC15, matsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] C15;
    delete[] uC15;

    auto * C33 = new float[matsize]();
    auto * uC33 = new uintc[matsize]();
    import_binary_float(Cijkl_folder + "C33.bin", Cij, nPoints);
    expand_boundary(Cij, C33);
    compression(C33, uC33, matsize, maxC33, minC33);    
    hipMalloc((void**)&(d_C33), matsize*sizeof(uintc));
    hipMemcpy(d_C33, uC33, matsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] C33;
    delete[] uC33;
    
    auto * C35 = new float[matsize]();
    auto * uC35 = new uintc[matsize]();
    import_binary_float(Cijkl_folder + "C35.bin", Cij, nPoints);
    expand_boundary(Cij, C35);
    compression(C35, uC35, matsize, maxC35, minC35);    
    hipMalloc((void**)&(d_C35), matsize*sizeof(uintc));
    hipMemcpy(d_C35, uC35, matsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] C35;
    delete[] uC35;

    auto * C55 = new float[matsize]();
    auto * uC55 = new uintc[matsize]();
    import_binary_float(Cijkl_folder + "C55.bin", Cij, nPoints);
    expand_boundary(Cij, C55);
    compression(C55, uC55, matsize, maxC55, minC55);    
    hipMalloc((void**)&(d_C55), matsize*sizeof(uintc));
    hipMemcpy(d_C55, uC55, matsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] C55;
    delete[] uC55;

    delete[] Cij;
}

void Elastic_ANI::initialization()
{
    float beta = 5.0f;

    sx = geometry->xsrc[geometry->sInd[srcId]];
    sz = geometry->zsrc[geometry->sInd[srcId]];

    sIdx = (int)((sx + 0.5f*dx) / dx);
    sIdz = (int)((sz + 0.5f*dz) / dz);

    float * h_skw = new float[DGS*DGS]();

    auto sgw = gaussian_weights(sx, sz, sIdx, sIdz, dx, dz);

    for (int zId = 0; zId < DGS; zId++)
        for (int xId = 0; xId < DGS; xId++)
            h_skw[zId + xId*DGS] = sgw[zId][xId];

    sIdx += nb; 
    sIdz += nb;

    int * h_rIdx = new int[max_spread]();
    int * h_rIdz = new int[max_spread]();

    float * h_rkwPs = new float[DGS*DGS*max_spread]();
    float * h_rkwVx = new float[DGS*DGS*max_spread]();
    float * h_rkwVz = new float[DGS*DGS*max_spread]();

    int spreadId = 0;

    for (recId = geometry->iRec[srcId]; recId < geometry->fRec[srcId]; recId++)
    {
        float rx = geometry->xrec[recId];
        float rz = geometry->zrec[recId];
        
        int rIdx = (int)((rx + 0.5f*dx) / dx);
        int rIdz = (int)((rz + 0.5f*dz) / dz);

        auto rkwPs = kaiser_weights(rx, rz, rIdx, rIdz, dx, dz, beta);        
        auto rkwVx = kaiser_weights(rx + 0.5f*dx, rz + 0.5f*dz, rIdx, rIdz, dx, dz, beta);
        auto rkwVz = kaiser_weights(rx + 0.5f*dx, rz + 0.5f*dz, rIdx, rIdz, dx, dz, beta);
        
        for (int zId = 0; zId < DGS; zId++)
        {
            for (int xId = 0; xId < DGS; xId++)
            {
                h_rkwPs[zId + xId*DGS + spreadId*DGS*DGS] = rkwPs[zId][xId];
                h_rkwVx[zId + xId*DGS + spreadId*DGS*DGS] = rkwVx[zId][xId];
                h_rkwVz[zId + xId*DGS + spreadId*DGS*DGS] = rkwVz[zId][xId];
            }
        }

        h_rIdx[spreadId] = rIdx + nb;
        h_rIdz[spreadId] = rIdz + nb;

        ++spreadId;
    }

    hipMemcpy(d_skw, h_skw, DGS*DGS*sizeof(float), hipMemcpyHostToDevice);
    
    hipMemcpy(d_rkwPs, h_rkwPs, DGS*DGS*max_spread*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rkwVx, h_rkwVx, DGS*DGS*max_spread*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rkwVz, h_rkwVz, DGS*DGS*max_spread*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_rIdx, h_rIdx, max_spread*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rIdz, h_rIdz, max_spread*sizeof(int), hipMemcpyHostToDevice);

    delete[] h_skw;
    delete[] h_rkwPs;
    delete[] h_rkwVx;
    delete[] h_rkwVz;
    delete[] h_rIdx;
    delete[] h_rIdz;
}

void Elastic_ANI::compute_eikonal()
{
    dim3 grid(1,1,1);
    dim3 block(MESHDIM+1,MESHDIM+1,1);

    time_set<<<nBlocks,NTHREADS>>>(d_T, matsize);
    time_init<<<grid,block>>>(d_T,d_S,sx,sz,dx,dz,sIdx,sIdz,nzz,nb);
    eikonal_solver();

    get_quasi_slowness<<<nBlocks,NTHREADS>>>(d_T,d_S,dx,dz,sIdx,sIdz,nxx,nzz,nb,d_C11,d_C13,d_C15,d_C33,d_C35,d_C55,minC11,
                                             maxC11,minC13,maxC13,minC15,maxC15,minC33,maxC33,minC35,maxC35,minC55,maxC55);

    time_set<<<nBlocks,NTHREADS>>>(d_T, matsize);
    time_init<<<grid,block>>>(d_T,d_S,sx,sz,dx,dz,sIdx,sIdz,nzz,nb);
    eikonal_solver();

    hipMemcpy(d_S, S, matsize * sizeof(float), hipMemcpyHostToDevice);
}

void Elastic_ANI::compute_velocity()
{
    compute_velocity_rsg<<<nBlocks, NTHREADS>>>(d_Vx, d_Vz, d_Txx, d_Tzz, d_Txz, d_T, d_B, minB, maxB, d1D, d2D, 
                                                d_wavelet, d_skw, dx, dz, dt, timeId, tlag, sIdx, sIdz, nxx, nzz, nb, nt);
}

void Elastic_ANI::compute_pressure()
{
    compute_pressure_rsg<<<nBlocks, NTHREADS>>>(d_Vx, d_Vz, d_Txx, d_Tzz, d_Txz, d_P, d_T, d_C11, d_C13, d_C15, d_C33, 
                                                d_C35, d_C55, timeId, tlag, dx, dz, dt, nxx, nzz, minC11, maxC11, minC13, 
                                                maxC13, minC15, maxC15, minC33, maxC33, minC35, maxC35, minC55, maxC55);    
}

__global__ void get_quasi_slowness(float * T, float * S, float dx, float dz, int sIdx, int sIdz, int nxx, int nzz, 
                                   int nb, uintc * C11, uintc * C13, uintc * C15, uintc * C33, uintc * C35, uintc * C55, 
                                   float minC11, float maxC11, float minC13, float maxC13, float minC15, float maxC15, 
                                   float minC33, float maxC33, float minC35, float maxC35, float minC55, float maxC55)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i = (int)(index % nzz);
    int j = (int)(index / nzz);

    const int n = 2;
    const int v = 3;

    float p[n];
    float C[v*v];
    float Gv[n];

    if ((i >= nb) && (i < nzz-nb) && (j >= nb) && (j < nxx-nb))
    {
        if (!((i == sIdz) && (j == sIdx)))    
        {
            float dTz = 0.5f*(T[(i+1) + j*nzz] - T[(i-1) + j*nzz]) / dz;
            float dTx = 0.5f*(T[i + (j+1)*nzz] - T[i + (j-1)*nzz]) / dx;

            float norm = sqrtf(dTx*dTx + dTz*dTz);

            p[0] = dTx / norm;
            p[1] = dTz / norm;
            
            float c11 = (minC11 + (static_cast<float>(C11[index]) - 1.0f) * (maxC11 - minC11) / (COMPRESS - 1));
            float c13 = (minC13 + (static_cast<float>(C13[index]) - 1.0f) * (maxC13 - minC13) / (COMPRESS - 1));
            float c15 = (minC15 + (static_cast<float>(C15[index]) - 1.0f) * (maxC15 - minC15) / (COMPRESS - 1));

            float c33 = (minC33 + (static_cast<float>(C33[index]) - 1.0f) * (maxC33 - minC33) / (COMPRESS - 1));
            float c35 = (minC35 + (static_cast<float>(C35[index]) - 1.0f) * (maxC35 - minC35) / (COMPRESS - 1));

            float c55 = (minC55 + (static_cast<float>(C55[index]) - 1.0f) * (maxC55 - minC55) / (COMPRESS - 1));

            C[0+0*v] = c11; C[0+1*v] = c13; C[0+2*v] = c15;
            C[1+0*v] = c13; C[1+1*v] = c33; C[1+2*v] = c35;
            C[2+0*v] = c15; C[2+1*v] = c35; C[2+2*v] = c55;

            float Ro = c33*S[index]*S[index];    
            
            for (int indp = 0; indp < v*v; indp++)
                C[indp] = C[indp] / Ro / Ro;

            float Gxx = C[0+0*v]*p[0]*p[0] + C[2+2*v]*p[1]*p[1] + 2.0f*C[0+2*v]*p[0]*p[1];
            float Gzz = C[2+2*v]*p[0]*p[0] + C[1+1*v]*p[1]*p[1] + 2.0f*C[1+2*v]*p[0]*p[1];
            float Gxz = C[0+2*v]*p[0]*p[0] + C[1+2*v]*p[1]*p[1] + (C[0+1*v] + C[2+2*v])*p[0]*p[1]; 
            
            float coeff1 = Gxx + Gzz;
            float coeff2 = Gxx - Gzz;
            
            float det = sqrtf((coeff2 * coeff2) / 4.0f + Gxz * Gxz);

            Gv[0] = coeff1 / 2.0 + det;
            Gv[1] = coeff1 / 2.0 - det;
            
            if (Gv[0] < Gv[1]) {float aux = Gv[0]; Gv[0] = Gv[1]; Gv[1] = aux;} 

            S[index] = 1.0f / sqrtf(Gv[0] * Ro);
        }
    }
}

__global__ void compute_velocity_rsg(float * Vx, float * Vz, float * Txx, float * Tzz, float * Txz, float * T, uintc * B, float minB, 
                                     float maxB, float * damp1D, float * damp2D, float * wavelet, float * skw, float dx, float dz, 
                                     float dt, int tId, int tlag, int sIdx, int sIdz, int nxx, int nzz, int nb, int nt)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i = (int)(index % nzz);
    int j = (int)(index / nzz);

    if ((index == 0) && (tId < nt))
    {
        for (int i = 0; i < DGS; i++)
        {
            int zi = sIdz + i - 2;
            for (int j = 0; j < DGS; j++)
            {
                int xi = sIdx + j - 2;

                Txx[zi + xi*nzz] += skw[i + j*DGS]*wavelet[tId] / (dx*dz);
                Tzz[zi + xi*nzz] += skw[i + j*DGS]*wavelet[tId] / (dx*dz);
            }
        }
    }

    float d1_Txx = 0.0f; float d2_Txx = 0.0f;
    float d1_Tzz = 0.0f; float d2_Tzz = 0.0f;
    float d1_Txz = 0.0f; float d2_Txz = 0.0f;
 
    float FDM[] = {FDM4, -FDM3, FDM2, -FDM1};
    
    if ((T[index] < (float)(tId + tlag)*dt) && (index < nxx*nzz))
    {
        if((i >= 3) && (i < nzz-4) && (j >= 3) && (j < nxx-4)) 
        {   
            # pragma unroll 4 
            for (int rsg = 0; rsg < 4; rsg++)
            {
                d1_Txx += FDM[rsg]*(Txx[(i+rsg+1) + (j+rsg+1)*nzz] - Txx[(i-rsg) + (j-rsg)*nzz]);
                d1_Tzz += FDM[rsg]*(Tzz[(i+rsg+1) + (j+rsg+1)*nzz] - Tzz[(i-rsg) + (j-rsg)*nzz]);
                d1_Txz += FDM[rsg]*(Txz[(i+rsg+1) + (j+rsg+1)*nzz] - Txz[(i-rsg) + (j-rsg)*nzz]);

                d2_Txx += FDM[rsg]*(Txx[(i-rsg) + (j+rsg+1)*nzz] - Txx[(i+rsg+1) + (j-rsg)*nzz]);
                d2_Tzz += FDM[rsg]*(Tzz[(i-rsg) + (j+rsg+1)*nzz] - Tzz[(i+rsg+1) + (j-rsg)*nzz]);
                d2_Txz += FDM[rsg]*(Txz[(i-rsg) + (j+rsg+1)*nzz] - Txz[(i+rsg+1) + (j-rsg)*nzz]);
            }
        }
    
        float dTxx_dx = 0.5f*(d1_Txx + d2_Txx) / dx;
        float dTxz_dx = 0.5f*(d1_Txz + d2_Txz) / dx;

        float dTxz_dz = 0.5f*(d1_Txz - d2_Txz) / dz;
        float dTzz_dz = 0.5f*(d1_Tzz - d2_Tzz) / dz;

        float B00 = (minB + (static_cast<float>(B[i + j*nzz]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));
        float B10 = (minB + (static_cast<float>(B[i + (j+1)*nzz]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));;
        float B01 = (minB + (static_cast<float>(B[(i+1) + j*nzz]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));;
        float B11 = (minB + (static_cast<float>(B[(i+1) + (j+1)*nzz]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));;

        float Bxz = 0.25f*(B00 + B10 + B01 + B11);

        Vx[index] += dt*Bxz*(dTxx_dx + dTxz_dz); 
        Vz[index] += dt*Bxz*(dTxz_dx + dTzz_dz);    
        
    	float damper = get_boundary_damper(damp1D, damp2D, i, j, nxx, nzz, nb);

        Vx[index] *= damper;
        Vz[index] *= damper;

        Txx[index] *= damper;
        Tzz[index] *= damper;
        Txz[index] *= damper;
    }
}

__global__ void compute_pressure_rsg(float * Vx, float * Vz, float * Txx, float * Tzz, float * Txz, float * P, float * T, 
                                     uintc * C11, uintc * C13, uintc * C15, uintc * C33, uintc * C35, uintc * C55, int tId, 
                                     int tlag, float dx, float dz, float dt, int nxx, int nzz, float minC11, float maxC11, 
                                     float minC13, float maxC13, float minC15, float maxC15, float minC33, float maxC33, 
                                     float minC35, float maxC35, float minC55, float maxC55)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i = (int)(index % nzz);
    int j = (int)(index / nzz);

    float d1_Vx = 0.0f; float d2_Vx = 0.0f;
    float d1_Vz = 0.0f; float d2_Vz = 0.0f;

    float FDM[] = {FDM4, -FDM3, FDM2, -FDM1};

    if ((T[index] < (float)(tId + tlag)*dt) && (index < nxx*nzz))
    {
        if((i > 3) && (i < nzz-3) && (j > 3) && (j < nxx-3)) 
        {
            # pragma unroll 4
            for (int rsg = 0; rsg < 4; rsg++)
            {       
                d1_Vx += FDM[rsg]*(Vx[(i+rsg) + (j+rsg)*nzz] - Vx[(i-rsg-1) + (j-rsg-1)*nzz]);      
                d1_Vz += FDM[rsg]*(Vz[(i+rsg) + (j+rsg)*nzz] - Vz[(i-rsg-1) + (j-rsg-1)*nzz]);      
    
                d2_Vx += FDM[rsg]*(Vx[(i-rsg-1) + (j+rsg)*nzz] - Vx[(i+rsg) + (j-rsg-1)*nzz]);      
                d2_Vz += FDM[rsg]*(Vz[(i-rsg-1) + (j+rsg)*nzz] - Vz[(i+rsg) + (j-rsg-1)*nzz]);      
            }
        }
    
        float dVx_dx = 0.5f*(d1_Vx + d2_Vx) / dx;
        float dVz_dx = 0.5f*(d1_Vz + d2_Vz) / dx;
        
        float dVx_dz = 0.5f*(d1_Vx - d2_Vx) / dz;
        float dVz_dz = 0.5f*(d1_Vz - d2_Vz) / dz;

        float c11 = (minC11 + (static_cast<float>(C11[index]) - 1.0f) * (maxC11 - minC11) / (COMPRESS - 1));
        float c13 = (minC13 + (static_cast<float>(C13[index]) - 1.0f) * (maxC13 - minC13) / (COMPRESS - 1));
        float c15 = (minC15 + (static_cast<float>(C15[index]) - 1.0f) * (maxC15 - minC15) / (COMPRESS - 1));
        float c33 = (minC33 + (static_cast<float>(C33[index]) - 1.0f) * (maxC33 - minC33) / (COMPRESS - 1));
        float c35 = (minC35 + (static_cast<float>(C35[index]) - 1.0f) * (maxC35 - minC35) / (COMPRESS - 1));    
        float c55 = (minC55 + (static_cast<float>(C55[index]) - 1.0f) * (maxC55 - minC55) / (COMPRESS - 1));
                
        Txx[index] += dt*(c11*dVx_dx + c13*dVz_dz + c15*(dVx_dz + dVz_dx));
        Tzz[index] += dt*(c13*dVx_dx + c33*dVz_dz + c35*(dVx_dz + dVz_dx));
        Txz[index] += dt*(c15*dVx_dx + c35*dVz_dz + c55*(dVx_dz + dVz_dx));
    
        if ((i > 3) && (i < nzz-4) && (j > 3) && (j < nxx-4))
        {
            P[index] = 0.5f*(Txx[index] + Tzz[index]);
        }
    }
}